
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define BLOCK_SIZE 512

#define _check(stmt)                                                      \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      printf("Failed to run stmt ", #stmt);                               \
      printf("Got CUDA error ...  ", hipGetErrorString(err));            \
      return -1;                                                          \
    }                                                                     \
  } while (0)

int main(int argc, char** argv) {
  if(argc != 3) {
    printf("Usage: ./scan <input_data_1> <input_data_2>\n");
    return -1;
  }

  return 0;
} 
